#include "hip/hip_runtime.h"
// peer_cutlass_enhanced_production.cu
// Production-ready PEER implementation with all correctness fixes

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <cuda/barrier>
#include <cuda/pipeline>
#include <cassert>
#include <hip/hip_bf16.h>
#include <mma.h>
#include <hiprand/hiprand_kernel.h>

// CUTLASS includes for optimized GEMM
#include <cutlass/cutlass.h>
#include <cutlass/gemm/device/gemm.h>
#include <cutlass/gemm/device/gemm_universal.h>
#include <cutlass/arch/mma.h>
#include <cutlass/epilogue/thread/linear_combination.h>
#include <cutlass/epilogue/thread/linear_combination_relu.h>
#include <cute/tensor.hpp>
#include <cute/atom/mma_atom.hpp>

#include <memory>
#include <vector>
#include <thread>
#include <atomic>
#include <chrono>
#include <mutex>

namespace peer {

using namespace cute;

// Helper for alignment
template<int N>
__host__ __device__ constexpr size_t align_to(size_t x) {
    return (x + N - 1) / N * N;
}

// POD struct for device-side expert pointers with heat tracking
struct ExpertPtrDev {
    const half* host_u;
    const half* host_v; 
    const half* dev_u;
    const half* dev_v;
    int hbm_slot;
    bool is_hot;
    unsigned int heat;  // Larger counter for multi-warp updates
};

// Device-side helper to fetch expert pointers
__device__ inline void fetch_expert(int id, ExpertPtrDev* experts,
                                   const half*& u, const half*& v) {
    auto& e = experts[id];
    u = e.is_hot ? e.dev_u : e.host_u;
    v = e.is_hot ? e.dev_v : e.host_v;
    
    // FIX 3: Update heat counter from all warps
    // Each warp's lane 0 increments
    if (threadIdx.x % 32 == 0) {
        atomicAdd(&e.heat, 1);
    }
}

// ======================== HIERARCHICAL MEMORY MANAGER ========================

class HierarchicalExpertCache {
private:
    // Expert metadata
    struct ExpertInfo {
        void* host_u_ptr;      // Pointer in system RAM
        void* host_v_ptr;
        void* device_u_ptr;    // Pointer in HBM (if cached)
        void* device_v_ptr;
        std::atomic<int> access_count{0};  // FIX: Make atomic to avoid races
        bool is_hot{false};
        int last_access_time{0};
        int hbm_slot{-1};      // Which HBM slot this expert occupies
    };
    
    std::vector<ExpertInfo> experts_;
    
    // CLOCK-based eviction (O(1) instead of O(N))
    std::vector<int> clock_hand_;  // Maps slot -> expert_id
    int clock_position_{0};
    
    // Memory pools
    void* hbm_pool_u_;
    void* hbm_pool_v_;
    size_t hbm_capacity_;  // Number of experts that fit in HBM
    size_t expert_u_bytes_;
    size_t expert_v_bytes_;
    
    // Prefetch thread
    std::thread prefetch_thread_;
    std::atomic<bool> should_stop_{false};
    std::mutex promotion_mutex_;  // Protect promotion decisions
    
    // Profiling - Split 128-bit atomic into two 64-bit
    std::atomic<int> cache_hits_{0};
    std::atomic<int> cache_misses_{0};
    std::atomic<uint64_t> bytes_lo_{0};
    std::atomic<uint64_t> bytes_hi_{0};
    
    // GPU device ID
    int device_id_;
    
    // Device-side expert pointer tables
    ExpertPtrDev* d_experts_managed_;  // Managed memory version
    ExpertPtrDev* d_experts_device_;   // Device-only mirror for perf
    
    // FIX 4: Delta tracking for heat sync
    unsigned int* d_heat_deltas_;      // Device array for heat deltas
    unsigned int* h_heat_deltas_;      // Host pinned buffer
    int num_experts_;
    
public:
    HierarchicalExpertCache(
        int num_experts,
        int input_dim,
        int hidden_dim,
        int output_dim,
        size_t hbm_capacity_mb = 16384  // 16GB for expert cache
    ) : num_experts_(num_experts) {
        expert_u_bytes_ = input_dim * hidden_dim * sizeof(half);
        expert_v_bytes_ = hidden_dim * output_dim * sizeof(half);
        __uint128_t bytes_per_expert = expert_u_bytes_ + expert_v_bytes_;
        
        hipGetDevice(&device_id_);
        
        // Calculate how many experts fit in HBM budget
        __uint128_t hbm_bytes = __uint128_t(hbm_capacity_mb) * 1024 * 1024;
        hbm_capacity_ = hbm_bytes / bytes_per_expert;
        hbm_capacity_ = std::min(hbm_capacity_, size_t(num_experts / 10));  // Max 10% in HBM
        
        printf("Hierarchical cache: %zu experts in HBM, %d total\n", 
               hbm_capacity_, num_experts);
        
        // Allocate HBM pool
        hipMalloc(&hbm_pool_u_, hbm_capacity_ * expert_u_bytes_);
        hipMalloc(&hbm_pool_v_, hbm_capacity_ * expert_v_bytes_);
        
        // Initialize expert metadata
        experts_.resize(num_experts);
        clock_hand_.resize(hbm_capacity_, -1);
        
        // Allocate device-side expert pointer tables
        hipMallocManaged(&d_experts_managed_, num_experts * sizeof(ExpertPtrDev));
        hipMalloc(&d_experts_device_, num_experts * sizeof(ExpertPtrDev));  // Device-only mirror
        
        // FIX 4: Allocate heat delta tracking
        hipMalloc(&d_heat_deltas_, num_experts * sizeof(unsigned int));
        hipHostMalloc(&h_heat_deltas_, num_experts * sizeof(unsigned int));
        hipMemset(d_heat_deltas_, 0, num_experts * sizeof(unsigned int));
        
        // Start prefetch thread
        prefetch_thread_ = std::thread(&HierarchicalExpertCache::prefetch_loop, this);
    }
    
    ~HierarchicalExpertCache() {
        should_stop_ = true;
        if (prefetch_thread_.joinable()) {
            prefetch_thread_.join();
        }
        hipFree(hbm_pool_u_);
        hipFree(hbm_pool_v_);
        hipFree(d_experts_managed_);
        hipFree(d_experts_device_);
        hipFree(d_heat_deltas_);
        hipHostFree(h_heat_deltas_);
    }
    
    // Allocate expert weights in system RAM using pinned memory
    void allocate_expert_weights(half* u_weights, half* v_weights, bool use_managed = false) {
        // Use pinned or managed memory based on flag
        for (int i = 0; i < experts_.size(); i++) {
            experts_[i].host_u_ptr = u_weights + i * (expert_u_bytes_ / sizeof(half));
            experts_[i].host_v_ptr = v_weights + i * (expert_v_bytes_ / sizeof(half));
            experts_[i].device_u_ptr = nullptr;
            experts_[i].device_v_ptr = nullptr;
            
            // Initialize device-side pointer table
            d_experts_managed_[i] = {
                (const half*)experts_[i].host_u_ptr,
                (const half*)experts_[i].host_v_ptr,
                nullptr,
                nullptr,
                -1,
                false,
                0  // Initial heat
            };
        }
        
        // Copy to device-only mirror for performance
        hipMemcpy(d_experts_device_, d_experts_managed_, 
                   num_experts_ * sizeof(ExpertPtrDev), hipMemcpyHostToDevice);
        
        if (use_managed) {
            // Proper size computation to avoid overflow
            size_t u_total_bytes = size_t(__uint128_t(experts_.size()) * expert_u_bytes_);
            size_t v_total_bytes = size_t(__uint128_t(experts_.size()) * expert_v_bytes_);
            
            // For managed memory, use advise
            hipMemAdvise(u_weights, u_total_bytes, 
                          hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
            hipMemAdvise(v_weights, v_total_bytes,
                          hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
            
            // Advise that GPU will access these pages
            hipMemAdvise(u_weights, u_total_bytes, 
                          hipMemAdviseSetAccessedBy, device_id_);
            hipMemAdvise(v_weights, v_total_bytes,
                          hipMemAdviseSetAccessedBy, device_id_);
            
            // Mark as read-mostly if CUDA 12+
            #if CUDA_VERSION >= 12000
            hipMemAdvise(u_weights, u_total_bytes, 
                          hipMemAdviseSetReadMostly, device_id_);
            hipMemAdvise(v_weights, v_total_bytes,
                          hipMemAdviseSetReadMostly, device_id_);
            #endif
        }
    }
    
    // Get device-side expert pointer table (use device mirror for perf)
    ExpertPtrDev* get_device_experts() const {
        return d_experts_device_;
    }
    
    // Host-side version - with stats and promotion
    __host__ void get_expert_ptrs(int expert_id, void*& u_ptr, void*& v_ptr, hipStream_t stream) {
        auto& info = experts_[expert_id];
        info.access_count.fetch_add(1);
        
        if (info.is_hot && info.device_u_ptr != nullptr) {
            // Expert is in HBM cache
            u_ptr = info.device_u_ptr;
            v_ptr = info.device_v_ptr;
            cache_hits_++;
        } else {
            // Expert in system RAM - will be accessed via UVA
            u_ptr = info.host_u_ptr;
            v_ptr = info.host_v_ptr;
            cache_misses_++;
            
            // Schedule for promotion if accessed frequently
            if (info.access_count.load() > 10 && !info.is_hot) {
                schedule_promotion(expert_id, stream);
            }
        }
    }
    
    // Turn the "hint" into a real prefetch
    void hint_future_access(const int* expert_ids, int count, hipStream_t stream) {
        for (int i = 0; i < count; i++) {
            int expert_id = expert_ids[i];
            auto& info = experts_[expert_id];
            
            if (!info.is_hot) {
                // Prefetch to GPU if not already there
                hipMemPrefetchAsync(info.host_u_ptr, expert_u_bytes_, 
                                     device_id_, stream);
                hipMemPrefetchAsync(info.host_v_ptr, expert_v_bytes_, 
                                     device_id_, stream);
                
                // Track bytes transferred using helper
                add_bytes(expert_u_bytes_ + expert_v_bytes_);
            }
        }
    }
    
    // FIX 4: Efficient heat counter sync using deltas
    void sync_heat_counters(hipStream_t stream) {
        // Extract heat deltas from device
        static __global__ void extract_heat_deltas_kernel(
            ExpertPtrDev* experts, unsigned int* deltas, int num_experts) {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if (idx < num_experts) {
                unsigned int heat = experts[idx].heat;
                if (heat > 0) {
                    deltas[idx] = heat;
                    experts[idx].heat = 0;  // Reset
                }
            }
        }
        
        // Run kernel to extract deltas
        int threads = 256;
        int blocks = (num_experts_ + threads - 1) / threads;
        extract_heat_deltas_kernel<<<blocks, threads, 0, stream>>>(
            d_experts_device_, d_heat_deltas_, num_experts_);
        
        // Copy only non-zero deltas
        hipMemcpyAsync(h_heat_deltas_, d_heat_deltas_,
                        num_experts_ * sizeof(unsigned int), 
                        hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);
        
        // Update host-side access counts
        for (int i = 0; i < num_experts_; i++) {
            if (h_heat_deltas_[i] > 0) {
                experts_[i].access_count.fetch_add(h_heat_deltas_[i]);
                h_heat_deltas_[i] = 0;  // Reset
            }
        }
        
        // Clear device deltas
        hipMemsetAsync(d_heat_deltas_, 0, num_experts_ * sizeof(unsigned int), stream);
    }
    
    // Implement print_stats
    void print_stats() {
        int hits = cache_hits_.load();
        int misses = cache_misses_.load();
        uint64_t lo = bytes_lo_.load();
        uint64_t hi = bytes_hi_.load();
        __uint128_t bytes = (__uint128_t(hi) << 64) | lo;
        
        if (hits + misses > 0) {
            float hit_rate = 100.0f * hits / (hits + misses);
            double gb_transferred = double(bytes) / (1024.0 * 1024.0 * 1024.0);
            
            printf("HierarchicalExpertCache Statistics:\n");
            printf("  Cache hit rate: %.1f%% (%d hits, %d misses)\n", 
                   hit_rate, hits, misses);
            printf("  Total data transferred: %.2f GB\n", gb_transferred);
            printf("  Hot experts: %d / %zu capacity\n", count_hot_experts(), hbm_capacity_);
        }
    }
    
private:
    // Helper to add bytes with 128-bit counter
    inline void add_bytes(uint64_t n) {
        uint64_t old = bytes_lo_.fetch_add(n, std::memory_order_relaxed);
        if (old > UINT64_MAX - n) {
            bytes_hi_.fetch_add(1, std::memory_order_relaxed);
        }
    }
    
    int count_hot_experts() {
        int count = 0;
        for (const auto& e : experts_) {
            if (e.is_hot) count++;
        }
        return count;
    }
    
    // O(1) CLOCK-based eviction with heat awareness
    void schedule_promotion(int expert_id, hipStream_t stream) {
        std::lock_guard<std::mutex> lock(promotion_mutex_);
        
        if (experts_[expert_id].is_hot) return;
        
        // Find a slot using CLOCK algorithm
        int slot = -1;
        for (int i = 0; i < hbm_capacity_ * 2; i++) {
            int candidate_slot = clock_position_;
            clock_position_ = (clock_position_ + 1) % hbm_capacity_;
            
            if (clock_hand_[candidate_slot] == -1) {
                // Empty slot
                slot = candidate_slot;
                break;
            }
            
            int victim_id = clock_hand_[candidate_slot];
            
            // Check both host-side access count and device-side heat
            bool has_activity = experts_[victim_id].access_count.load() > 0 ||
                               (d_experts_managed_[victim_id].heat > 128);  // High heat threshold
            
            if (!has_activity) {
                // Found victim
                evict_expert(victim_id);
                slot = candidate_slot;
                break;
            } else {
                // Give second chance
                experts_[victim_id].access_count.store(0);
                d_experts_managed_[victim_id].heat /= 2;  // Decay heat
            }
        }
        
        if (slot >= 0) {
            promote_expert(expert_id, slot, stream);
        }
    }
    
    void promote_expert(int expert_id, int slot, hipStream_t stream) {
        experts_[expert_id].hbm_slot = slot;
        clock_hand_[slot] = expert_id;
        
        // Copy to HBM
        experts_[expert_id].device_u_ptr = (char*)hbm_pool_u_ + slot * expert_u_bytes_;
        experts_[expert_id].device_v_ptr = (char*)hbm_pool_v_ + slot * expert_v_bytes_;
        
        hipMemcpyAsync(experts_[expert_id].device_u_ptr,
                        experts_[expert_id].host_u_ptr,
                        expert_u_bytes_, hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(experts_[expert_id].device_v_ptr,
                        experts_[expert_id].host_v_ptr,
                        expert_v_bytes_, hipMemcpyHostToDevice, stream);
        
        experts_[expert_id].is_hot = true;
        
        // Update device-side pointer table
        d_experts_managed_[expert_id].dev_u = (const half*)experts_[expert_id].device_u_ptr;
        d_experts_managed_[expert_id].dev_v = (const half*)experts_[expert_id].device_v_ptr;
        d_experts_managed_[expert_id].hbm_slot = slot;
        d_experts_managed_[expert_id].is_hot = true;
        
        // Update device mirror asynchronously
        hipMemcpyAsync(d_experts_device_ + expert_id, d_experts_managed_ + expert_id,
                        sizeof(ExpertPtrDev), hipMemcpyHostToDevice, stream);
        
        add_bytes(expert_u_bytes_ + expert_v_bytes_);
    }
    
    void evict_expert(int expert_id) {
        int slot = experts_[expert_id].hbm_slot;
        if (slot >= 0) {
            clock_hand_[slot] = -1;
        }
        
        experts_[expert_id].is_hot = false;
        experts_[expert_id].device_u_ptr = nullptr;
        experts_[expert_id].device_v_ptr = nullptr;
        experts_[expert_id].hbm_slot = -1;
        
        // Update device-side pointer table
        d_experts_managed_[expert_id].dev_u = nullptr;
        d_experts_managed_[expert_id].dev_v = nullptr;
        d_experts_managed_[expert_id].hbm_slot = -1;
        d_experts_managed_[expert_id].is_hot = false;
    }
    
    void prefetch_loop() {
        hipStream_t prefetch_stream;
        hipStreamCreate(&prefetch_stream);
        
        while (!should_stop_) {
            std::this_thread::sleep_for(std::chrono::milliseconds(100));
            
            static int counter = 0;
            counter++;
            
            // Sync heat counters periodically
            if (counter % 10 == 0) {
                sync_heat_counters(prefetch_stream);
            }
            
            // Print stats periodically
            if (counter % 50 == 0) {
                print_stats();
            }
        }
        
        hipStreamDestroy(prefetch_stream);
    }
};

// ======================== OPTIMIZED GEMM USING CUTLASS ========================

// CUTLASS-based GEMM for optimal performance
template<typename Element>
struct OptimizedGemm {
    // Define the GEMM operation
    using ElementA = Element;
    using ElementB = Element;
    using ElementC = Element;
    using ElementAccumulator = float;
    
    using ThreadblockShape = cutlass::gemm::GemmShape<16, 128, 32>;
    using WarpShape = cutlass::gemm::GemmShape<16, 32, 32>;
    using InstructionShape = cutlass::gemm::GemmShape<16, 8, 16>;
    
    using EpilogueOp = cutlass::epilogue::thread::LinearCombination<
        ElementC,
        128 / cutlass::sizeof_bits<ElementC>::value,
        ElementAccumulator,
        ElementAccumulator
    >;
    
    using Gemm = cutlass::gemm::device::GemmUniversal<
        ElementA, cutlass::layout::RowMajor,
        ElementB, cutlass::layout::RowMajor,
        ElementC, cutlass::layout::RowMajor,
        ElementAccumulator,
        cutlass::arch::OpClassTensorOp,
        cutlass::arch::Sm90,
        ThreadblockShape,
        WarpShape,
        InstructionShape,
        EpilogueOp,
        cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
        4  // Stages
    >;
    
    // FIX 2: WMMA with proper bounds checking
    __device__ static void gemm_tn_safe(
        const Element* A,  // M x K (row-major)
        const Element* B,  // N x K (row-major, so B^T is K x N)
        Element* C,        // M x N (row-major) 
        float scale,
        int M, int N, int K
    ) {
        using namespace nvcuda::wmma;
        
        constexpr int WMMA_M = 16;
        constexpr int WMMA_N = 16;
        constexpr int WMMA_K = 16;
        
        // Check if dimensions are suitable for WMMA
        if (M < WMMA_M || N < WMMA_N || K < WMMA_K) {
            // Fallback to simple loop for small matrices
            const int tid = threadIdx.x;
            for (int idx = tid; idx < M * N; idx += blockDim.x) {
                int m = idx / N;
                int n = idx % N;
                float acc = 0.0f;
                for (int k = 0; k < K; k++) {
                    acc += float(A[m * K + k]) * float(B[n * K + k]);
                }
                C[idx] = Element(acc * scale);
            }
            return;
        }
        
        // WMMA path for larger matrices
        fragment<matrix_a, WMMA_M, WMMA_N, WMMA_K, __half, row_major> a_frag;
        fragment<matrix_b, WMMA_M, WMMA_N, WMMA_K, __half, col_major> b_frag;
        fragment<accumulator, WMMA_M, WMMA_N, WMMA_K, __half> c_frag;
        
        const int warp_id = threadIdx.x / 32;
        const int num_warps = blockDim.x / 32;
        
        // Process tiles with bounds checking
        for (int m = warp_id * WMMA_M; m < M; m += num_warps * WMMA_M) {
            for (int n = 0; n < N; n += WMMA_N) {
                // FIX 2: Proper bounds check
                if (m + WMMA_M > M || n + WMMA_N > N) continue;
                
                fill_fragment(c_frag, __float2half(0.0f));
                
                for (int k = 0; k < K; k += WMMA_K) {
                    // Bounds check for K dimension
                    if (k + WMMA_K > K) continue;
                    
                    // Load A fragment
                    load_matrix_sync(a_frag, A + m * K + k, K);
                    
                    // Load B fragment (transposed)
                    load_matrix_sync(b_frag, B + n * K + k, K);
                    
                    // Compute
                    mma_sync(c_frag, a_frag, b_frag, c_frag);
                }
                
                // Scale and store
                if (scale != 1.0f) {
                    for (int i = 0; i < c_frag.num_elements; i++) {
                        c_frag.x[i] = __float2half(float(c_frag.x[i]) * scale);
                    }
                }
                
                // Store result
                store_matrix_sync(C + m * N + n, c_frag, N, mem_row_major);
            }
        }
    }
};

// ======================== PRODUCT KEY ROUTING ========================

template<typename scalar_t, int top_k, int sqrt_n>
__device__ void product_key_routing(
    const scalar_t* query,      // [d]
    const scalar_t* sub_keys1,  // [sqrt_n, d]
    const scalar_t* sub_keys2,  // [sqrt_n, d]
    int d,
    int* expert_indices,        // [top_k]
    float* expert_scores,       // [top_k]
    bool norm_keys = true,
    bool norm_query = true
) {
    float scores1[sqrt_n];
    float scores2[sqrt_n];
    
    // Normalize query if requested
    float query_norm = 0.0f;
    if (norm_query) {
        for (int i = 0; i < d; i++) {
            query_norm += float(query[i]) * float(query[i]);
        }
        query_norm = rsqrtf(query_norm + 1e-6f);
    }
    
    // Compute scores with first set of sub-keys
    for (int i = 0; i < sqrt_n; i++) {
        float score = 0.0f;
        float key_norm = 0.0f;
        
        for (int j = 0; j < d; j++) {
            float q_val = float(query[j]);
            if (norm_query) q_val *= query_norm;
            
            float k_val = float(sub_keys1[i * d + j]);
            score += q_val * k_val;
            
            if (norm_keys) {
                key_norm += k_val * k_val;
            }
        }
        
        if (norm_keys) {
            key_norm = rsqrtf(key_norm + 1e-6f);
            score *= key_norm;
        }
        
        scores1[i] = score;
    }
    
    // Compute scores with second set of sub-keys
    for (int i = 0; i < sqrt_n; i++) {
        float score = 0.0f;
        float key_norm = 0.0f;
        
        for (int j = 0; j < d; j++) {
            float q_val = float(query[j]);
            if (norm_query) q_val *= query_norm;
            
            float k_val = float(sub_keys2[i * d + j]);
            score += q_val * k_val;
            
            if (norm_keys) {
                key_norm += k_val * k_val;
            }
        }
        
        if (norm_keys) {
            key_norm = rsqrtf(key_norm + 1e-6f);
            score *= key_norm;
        }
        
        scores2[i] = score;
    }
    
    // Find top-k product scores
    struct Score {
        float value;
        int index;
    };
    
    Score top_scores[top_k];
    for (int i = 0; i < top_k; i++) {
        top_scores[i].value = -1e10f;
        top_scores[i].index = -1;
    }
    
    // Compute all product scores and maintain top-k
    for (int i = 0; i < sqrt_n; i++) {
        for (int j = 0; j < sqrt_n; j++) {
            float prod_score = scores1[i] * scores2[j];
            int expert_id = i * sqrt_n + j;
            
            // Insert into top-k if necessary
            if (prod_score > top_scores[top_k-1].value) {
                // Find insertion point
                int pos = top_k - 1;
                while (pos > 0 && prod_score > top_scores[pos-1].value) {
                    pos--;
                }
                
                // Shift and insert
                for (int k = top_k - 1; k > pos; k--) {
                    top_scores[k] = top_scores[k-1];
                }
                top_scores[pos].value = prod_score;
                top_scores[pos].index = expert_id;
            }
        }
    }
    
    // Apply softmax to top-k scores
    float sum_exp = 0.0f;
    for (int i = 0; i < top_k; i++) {
        top_scores[i].value = expf(top_scores[i].value);
        sum_exp += top_scores[i].value;
    }
    
    // Write output
    for (int i = 0; i < top_k; i++) {
        expert_indices[i] = top_scores[i].index;
        expert_scores[i] = top_scores[i].value / sum_exp;
    }
}

// ======================== ENHANCED KERNEL WITH L2 OPTIMIZATION ========================

// Runtime L2 chunk size calculation
template<typename T>
__host__ __device__ inline int compute_l2_chunk_size(int input_dim) {
    constexpr int L2_SIZE_BYTES = 40 * 1024 * 1024;  // 40MB L2 on H100
    int bytes_per_token = input_dim * sizeof(T);
    return L2_SIZE_BYTES / bytes_per_token;
}

// PEERConfig struct definition
template<int MaxExperts_, int BlockM_, int BlockK_, int HiddenSize_, int OuterTiles_>
struct PEERConfig {
    static constexpr int MaxExperts = MaxExperts_;
    static constexpr int BlockM = BlockM_;
    static constexpr int BlockK = BlockK_;
    static constexpr int HiddenSize = HiddenSize_;
    static constexpr int OuterTiles = OuterTiles_;
};

template<
    typename Config,
    typename Element,
    int NumHeads,
    int TopK,
    int QueryDim,
    int SqrtN,
    int OUT,
    int BLOCK_DIM
>
__global__ void peer_kernel_enhanced(
    const Element* __restrict__ input,
    const Element* __restrict__ query_weight,
    const Element* __restrict__ query_bias,
    const Element* __restrict__ sub_keys1,
    const Element* __restrict__ sub_keys2,
    Element* __restrict__ output,
    ExpertPtrDev* d_experts,  // Non-const for heat updates
    const Element* __restrict__ bn_scale,
    const Element* __restrict__ bn_bias,
    int B, int S, int IN,
    int chunk_size,  // Runtime parameter
    bool use_batch_norm = true,
    bool norm_keys = true,
    bool norm_query = true
) {
    extern __shared__ char smem_bytes[];
    
    // Compute proper shared memory layout with padding
    size_t tok_bytes = align_to<64>(chunk_size * IN * sizeof(Element));
    size_t u_bytes = align_to<64>(IN * Config::HiddenSize * sizeof(Element));
    size_t v_bytes = align_to<64>(Config::HiddenSize * OUT * sizeof(Element));
    
    // Shared memory layout with proper padding
    Element* token_cache = reinterpret_cast<Element*>(smem_bytes);
    Element* u_buffer[2];
    Element* v_buffer[2];
    u_buffer[0] = reinterpret_cast<Element*>((char*)token_cache + tok_bytes);
    u_buffer[1] = reinterpret_cast<Element*>((char*)u_buffer[0] + u_bytes);
    v_buffer[0] = reinterpret_cast<Element*>((char*)u_buffer[1] + u_bytes);
    v_buffer[1] = reinterpret_cast<Element*>((char*)v_buffer[0] + v_bytes);
    
    // Proper layout to avoid overlap
    Element* query_smem = reinterpret_cast<Element*>((char*)v_buffer[1] + v_bytes);
    size_t query_bytes = align_to<64>(QueryDim * sizeof(Element));
    Element* hidden_smem = reinterpret_cast<Element*>((char*)query_smem + query_bytes);
    
    // Pipeline for overlapping copy/compute
    __shared__ cuda::pipeline<cuda::thread_scope_block> pipe;
    auto pipe_role = cuda::make_pipeline_role(pipe);
    
    const int tid = threadIdx.x;
    const int warp_id = tid / 32;
    const int lane_id = tid % 32;
    
    // Process chunks of tokens
    for (int chunk_start = blockIdx.x * chunk_size; 
         chunk_start < B * S; 
         chunk_start += gridDim.x * chunk_size) {
        
        int chunk_end = min(chunk_start + chunk_size, B * S);
        int actual_chunk_size = chunk_end - chunk_start;
        
        // Stage 1: Load token chunk into shared memory
        for (int idx = tid; idx < actual_chunk_size * IN; idx += blockDim.x) {
            int token_offset = idx / IN;
            int feat_offset = idx % IN;
            int token_idx = chunk_start + token_offset;
            int batch_idx = token_idx / S;
            int seq_idx = token_idx % S;
            
            token_cache[idx] = input[batch_idx * S * IN + seq_idx * IN + feat_offset];
        }
        __syncthreads();
        
        // Process each token in the chunk
        for (int local_token_idx = 0; local_token_idx < actual_chunk_size; local_token_idx++) {
            Element* local_token = token_cache + local_token_idx * IN;
            int global_token_idx = chunk_start + local_token_idx;
            
            // Compute MAX_OUT_PER_THREAD based on actual block dim
            constexpr int OUT_PER_THREAD = (OUT + BLOCK_DIM - 1) / BLOCK_DIM;
            float thread_output[OUT_PER_THREAD];
            
            for (int i = 0; i < OUT_PER_THREAD && tid * OUT_PER_THREAD + i < OUT; i++) {
                thread_output[i] = 0.0f;
            }
            
            // Process each head
            for (int h = 0; h < NumHeads; h++) {
                // All warps cooperatively compute query projection
                OptimizedGemm<Element>::gemm_tn_safe(
                    local_token,
                    query_weight + h * QueryDim * IN,
                    query_smem,
                    1.0f,
                    1, QueryDim, IN
                );
                __syncthreads();
                
                // Add bias (distributed across threads)
                if (query_bias != nullptr && tid < QueryDim) {
                    query_smem[tid] = Element(float(query_smem[tid]) + 
                                             float(query_bias[h * QueryDim + tid]));
                }
                __syncthreads();
                
                // Batch normalization
                if (use_batch_norm && tid == 0) {
                    float mean = 0.0f, var = 0.0f;
                    for (int i = 0; i < QueryDim; i++) mean += float(query_smem[i]);
                    mean /= QueryDim;
                    for (int i = 0; i < QueryDim; i++) {
                        float diff = float(query_smem[i]) - mean;
                        var += diff * diff;
                    }
                    var = rsqrtf(var / QueryDim + 1e-5f);
                    for (int i = 0; i < QueryDim; i++) {
                        float normalized = (float(query_smem[i]) - mean) * var;
                        if (bn_scale != nullptr) {
                            normalized = normalized * float(bn_scale[h * QueryDim + i]) + 
                                        float(bn_bias[h * QueryDim + i]);
                        }
                        query_smem[i] = Element(normalized);
                    }
                }
                __syncthreads();
                
                // Product key routing
                __shared__ int expert_indices[TopK];
                __shared__ float expert_scores[TopK];
                
                if (tid == 0) {
                    product_key_routing<Element, TopK, SqrtN>(
                        query_smem, sub_keys1, sub_keys2, QueryDim,
                        expert_indices, expert_scores,
                        norm_keys, norm_query
                    );
                }
                __syncthreads();
                
                // Double-buffered expert computation with overlapped copy/compute
                int buffer_id = 0;
                
                for (int k = 0; k < TopK; k++) {
                    int expert_id = expert_indices[k];
                    float weight = expert_scores[k];
                    
                    // Get expert pointers (all warps participate)
                    const half *u_ptr = nullptr, *v_ptr = nullptr;
                    if (tid == 0) {
                        fetch_expert(expert_id, d_experts, u_ptr, v_ptr);
                    }
                    __syncthreads();
                    
                    // FIX 1: Proper 64-bit pointer broadcast
                    uint64_t u_addr = 0, v_addr = 0;
                    if (tid == 0) {
                        u_addr = reinterpret_cast<uint64_t>(u_ptr);
                        v_addr = reinterpret_cast<uint64_t>(v_ptr);
                    }
                    u_addr = __shfl_sync(0xffffffff, u_addr, 0);
                    v_addr = __shfl_sync(0xffffffff, v_addr, 0);
                    u_ptr = reinterpret_cast<const half*>(u_addr);
                    v_ptr = reinterpret_cast<const half*>(v_addr);
                    
                    // FIX 4: Load with 64-byte chunks
                    // Producer warps load while consumer warps compute
                    if (warp_id < 2) {
                        // Load U weights
                        if (warp_id == 0) {
                            size_t bytes = IN * Config::HiddenSize * sizeof(Element);
                            char* dst = (char*)u_buffer[buffer_id];
                            const char* src = (const char*)u_ptr;
                            
                            for (int off = tid * 64; off < bytes; off += BLOCK_DIM * 64) {
                                if (off + 64 <= bytes) {
                                    asm volatile("cp.async.bulk.shared::cluster.global [%0], [%1], 64;"
                                               :: "r"((uint32_t)__cvta_generic_to_shared(dst + off)), 
                                                  "l"(src + off));
                                }
                            }
                        }
                        // Load V weights
                        else {
                            size_t bytes = Config::HiddenSize * OUT * sizeof(Element);
                            char* dst = (char*)v_buffer[buffer_id];
                            const char* src = (const char*)v_ptr;
                            
                            for (int off = (tid - 32) * 64; off < bytes; off += (BLOCK_DIM - 32) * 64) {
                                if (off + 64 <= bytes) {
                                    asm volatile("cp.async.bulk.shared::cluster.global [%0], [%1], 64;"
                                               :: "r"((uint32_t)__cvta_generic_to_shared(dst + off)), 
                                                  "l"(src + off));
                                }
                            }
                        }
                    }
                    
                    // Commit and wait for copy
                    asm volatile("cp.async.commit_group;");
                    asm volatile("cp.async.wait_group 0;");
                    __syncthreads();
                    
                    // All warps compute GEMM
                    // U^T * x -> hidden
                    OptimizedGemm<Element>::gemm_tn_safe(
                        local_token,          // 1 x IN
                        u_buffer[buffer_id],  // HiddenSize x IN (transposed)
                        hidden_smem,
                        1.0f,
                        1, Config::HiddenSize, IN
                    );
                    __syncthreads();
                    
                    // Fused GELU in epilogue (distributed across threads)
                    for (int i = tid; i < Config::HiddenSize; i += blockDim.x) {
                        float x = float(hidden_smem[i]);
                        // GELU(x) = 0.5 * x * (1 + tanh(sqrt(2/pi) * (x + 0.044715 * x^3)))
                        float x3 = x * x * x;
                        float tanh_arg = 0.7978845608f * (x + 0.044715f * x3);
                        hidden_smem[i] = Element(0.5f * x * (1.0f + tanhf(tanh_arg)));
                    }
                    __syncthreads();
                    
                    // V * hidden -> output accumulation
                    for (int i = 0; i < OUT_PER_THREAD; i++) {
                        int out_idx = tid * OUT_PER_THREAD + i;
                        if (out_idx < OUT) {
                            float acc = 0.0f;
                            for (int j = 0; j < Config::HiddenSize; j++) {
                                acc += float(v_buffer[buffer_id][out_idx * Config::HiddenSize + j]) *
                                       float(hidden_smem[j]);
                            }
                            thread_output[i] += weight * acc;
                        }
                    }
                    
                    // Switch buffers
                    buffer_id = 1 - buffer_id;
                    __syncthreads();
                }
            } // End head loop
            
            // Write accumulated output once per token
            int batch_idx = global_token_idx / S;
            int seq_idx = global_token_idx % S;
            Element* out_ptr = output + batch_idx * S * OUT + seq_idx * OUT;
            
            for (int i = 0; i < OUT_PER_THREAD; i++) {
                int out_idx = tid * OUT_PER_THREAD + i;
                if (out_idx < OUT) {
                    // Direct write without atomics
                    out_ptr[out_idx] = Element(thread_output[i]);
                }
            }
            __syncthreads();
        }
    }
}

// ======================== C++ WRAPPER WITH UVA SUPPORT ========================

// Helper to set shared memory configuration
void set_smem_config(void* kernel_ptr, size_t smem_size) {
    hipError_t err = hipFuncSetAttribute(reinterpret_cast<const void*>(kernel_ptr), 
                                           hipFuncAttributeMaxDynamicSharedMemorySize, 
                                           smem_size);
    if (err != hipSuccess) {
        printf("Warning: Could not set shared memory size to %zu bytes: %s\n", 
               smem_size, hipGetErrorString(err));
        
        // Try setting carveout for more shared memory
        hipFuncSetAttribute(reinterpret_cast<const void*>(kernel_ptr),
                            hipFuncAttributePreferredSharedMemoryCarveout,
                            cudaSharedmemCarveoutMaxShared);
    }
}

class PEEROperatorEnhanced {
private:
    int num_experts_;
    int num_heads_;
    int top_k_;
    int query_dim_;
    int expert_hidden_size_;
    int sqrt_n_;
    int input_dim_;
    int output_dim_;
    
    // Hierarchical memory cache
    std::unique_ptr<HierarchicalExpertCache> cache_;
    
    // UVA-allocated weights
    half* u_weights_;
    half* v_weights_;
    bool use_managed_memory_;
    
public:
    PEEROperatorEnhanced(
        int num_experts,
        int num_heads,
        int top_k,
        int query_dim,
        int expert_hidden_size,
        int input_dim,
        int output_dim,
        size_t hbm_cache_mb = 16384,
        bool use_managed = false  // Option to use pinned memory
    ) : num_experts_(num_experts),
        num_heads_(num_heads),
        top_k_(top_k),
        query_dim_(query_dim),
        expert_hidden_size_(expert_hidden_size),
        sqrt_n_(int(std::sqrt(double(num_experts)) + 0.5)),  // Proper rounding
        input_dim_(input_dim),
        output_dim_(output_dim),
        u_weights_(nullptr),
        v_weights_(nullptr),
        use_managed_memory_(use_managed) {
        
        // Create hierarchical cache
        cache_ = std::make_unique<HierarchicalExpertCache>(
            num_experts, input_dim, expert_hidden_size, output_dim, hbm_cache_mb
        );
    }
    
    ~PEEROperatorEnhanced() {
        if (u_weights_) {
            if (use_managed_memory_) {
                hipFree(u_weights_);
            } else {
                hipHostFree(u_weights_);
            }
        }
        if (v_weights_) {
            if (use_managed_memory_) {
                hipFree(v_weights_);
            } else {
                hipHostFree(v_weights_);
            }
        }
    }
    
    void allocate_weights() {
        // Allocate using pinned or managed memory
        __uint128_t u_size = __uint128_t(num_experts_) * input_dim_ * expert_hidden_size_ * sizeof(half);
        __uint128_t v_size = __uint128_t(num_experts_) * expert_hidden_size_ * output_dim_ * sizeof(half);
        
        if (use_managed_memory_) {
            // Managed memory (slower first access)
            hipMallocManaged(&u_weights_, u_size);
            hipMallocManaged(&v_weights_, v_size);
        } else {
            // Use pinned memory for better performance
            hipHostMalloc(&u_weights_, u_size);
            hipHostMalloc(&v_weights_, v_size);
        }
        
        // Initialize with random values (in production, load from checkpoint)
        // ... initialization code ...
        
        // Register with cache
        cache_->allocate_expert_weights(u_weights_, v_weights_, use_managed_memory_);
        
        printf("Allocated %.2f GB of expert weights in %s memory\n",
               double(u_size + v_size) / (1024.0 * 1024.0 * 1024.0),
               use_managed_memory_ ? "managed" : "pinned");
    }
    
    void forward(
        const half* input,
        const half* query_weight,
        const half* query_bias,
        const half* sub_keys1,
        const half* sub_keys2,
        half* output,
        int batch_size,
        int seq_len,
        hipStream_t stream = 0
    ) {
        // Compute chunk size at runtime
        int chunk_size = compute_l2_chunk_size<half>(input_dim_);
        chunk_size = std::min(chunk_size, batch_size * seq_len);
        chunk_size = std::max(chunk_size, 1);  // At least 1 token
        
        // Enhanced kernel configuration
        using Config = PEERConfig<1048576, 56, 128, 256, 64>;
        constexpr int BLOCK_DIM = 128;
        
        int num_tokens = batch_size * seq_len;
        int grid_size = (num_tokens + chunk_size - 1) / chunk_size;
        grid_size = min(grid_size, 256);  // Limit grid size
        
        // Calculate shared memory with proper padding
        constexpr int TopK = 16;
        size_t smem_size = 0;
        smem_size += align_to<64>(chunk_size * input_dim_ * sizeof(half));  // Token cache
        smem_size += 2 * align_to<64>(input_dim_ * Config::HiddenSize * sizeof(half));  // U buffers
        smem_size += 2 * align_to<64>(Config::HiddenSize * output_dim_ * sizeof(half));  // V buffers
        smem_size += align_to<64>(query_dim_ * sizeof(half));  // Query (FP16)
        smem_size += TopK * Config::HiddenSize * sizeof(half);  // Hidden activations (FP16)
        
        // Set shared memory configuration
        auto kernel_func = peer_kernel_enhanced<Config, half, 8, TopK, 256, 1024, 1024, BLOCK_DIM>;
        set_smem_config((void*)kernel_func, smem_size);
        
        // No need to memset - kernel directly writes output
        
        // Launch enhanced kernel
        kernel_func<<<grid_size, BLOCK_DIM, smem_size, stream>>>(
            input, query_weight, query_bias,
            sub_keys1, sub_keys2, output,
            cache_->get_device_experts(),  // Device-only mirror
            nullptr, nullptr,  // bn_scale, bn_bias
            batch_size, seq_len, input_dim_,
            chunk_size,  // Runtime parameter
            true, true, true
        );
        
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            throw std::runtime_error(
                std::string("Enhanced PEER kernel launch failed: ") + 
                hipGetErrorString(err)
            );
        }
    }
    
    void print_cache_stats() {
        cache_->print_stats();
    }
    
    // Getters for validation in wrapper
    int num_experts() const { return num_experts_; }
    int num_heads() const { return num_heads_; }
    int input_dim() const { return input_dim_; }
    int output_dim() const { return output_dim_; }
};

// ======================== SMOKE TEST ========================

void smoke_test() {
    // Test configuration as suggested
    constexpr int B = 2;
    constexpr int S = 4;
    constexpr int IN = 64;
    constexpr int OUT = 128;
    constexpr int Hidden = 32;
    constexpr int Experts = 512;
    constexpr int NumHeads = 8;
    constexpr int TopK = 4;
    constexpr int QueryDim = 64;
    
    printf("Running smoke test: B=%d, S=%d, IN=%d, OUT=%d, Hidden=%d, Experts=%d\n",
           B, S, IN, OUT, Hidden, Experts);
    
    // Create operator
    PEEROperatorEnhanced op(Experts, NumHeads, TopK, QueryDim, Hidden, IN, OUT, 
                           1024, // 1GB HBM cache for testing
                           false); // Use pinned memory
    
    // Allocate weights
    op.allocate_weights();
    
    // Allocate test inputs/outputs
    half *d_input, *d_query_weight, *d_query_bias, *d_sub_keys1, *d_sub_keys2, *d_output;
    hipMalloc(&d_input, B * S * IN * sizeof(half));
    hipMalloc(&d_query_weight, NumHeads * QueryDim * IN * sizeof(half));
    hipMalloc(&d_query_bias, NumHeads * QueryDim * sizeof(half));
    hipMalloc(&d_sub_keys1, int(std::sqrt(Experts) + 0.5) * QueryDim * sizeof(half));
    hipMalloc(&d_sub_keys2, int(std::sqrt(Experts) + 0.5) * QueryDim * sizeof(half));
    hipMalloc(&d_output, B * S * OUT * sizeof(half));
    
    // Initialize with random data
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandGenerateUniform(gen, (float*)d_input, B * S * IN / 2);
    hiprandGenerateUniform(gen, (float*)d_query_weight, NumHeads * QueryDim * IN / 2);
    hiprandGenerateUniform(gen, (float*)d_query_bias, NumHeads * QueryDim / 2);
    hiprandGenerateUniform(gen, (float*)d_sub_keys1, int(std::sqrt(Experts) + 0.5) * QueryDim / 2);
    hiprandGenerateUniform(gen, (float*)d_sub_keys2, int(std::sqrt(Experts) + 0.5) * QueryDim / 2);
    hiprandDestroyGenerator(gen);
    
    // Run forward pass
    hipStream_t stream;
    hipStreamCreate(&stream);
    
    // Warmup
    for (int i = 0; i < 3; i++) {
        op.forward(d_input, d_query_weight, d_query_bias, d_sub_keys1, d_sub_keys2, 
                   d_output, B, S, stream);
    }
    hipStreamSynchronize(stream);
    
    // Time the kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start, stream);
    for (int i = 0; i < 10; i++) {
        op.forward(d_input, d_query_weight, d_query_bias, d_sub_keys1, d_sub_keys2, 
                   d_output, B, S, stream);
    }
    hipEventRecord(stop, stream);
    hipStreamSynchronize(stream);
    
    float ms;
    hipEventElapsedTime(&ms, start, stop);
    printf("Average kernel time: %.3f ms\n", ms / 10.0f);
    
    // Check for errors
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    } else {
        printf("Smoke test PASSED!\n");
    }
    
    // Print cache stats
    op.print_cache_stats();
    
    // Cleanup
    hipFree(d_input);
    hipFree(d_query_weight);
    hipFree(d_query_bias);
    hipFree(d_sub_keys1);
    hipFree(d_sub_keys2);
    hipFree(d_output);
    hipStreamDestroy(stream);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

}  // namespace peer

// For testing
#ifdef COMPILE_SMOKE_TEST
int main() {
    peer::smoke_test();
    return 0;
}
#endif